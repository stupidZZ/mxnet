#include "hip/hip_runtime.h"
/*!
* Copyright (c) 2017 by Contributors
* \file psroi_pooling.cu
* \brief psroi pooling operator
* \author Yi Li, Guodong Zhang
*/
#include "./psroi_pooling_align-inl.h"
#include <mshadow/tensor.h>
#include <mshadow/cuda/reduce.cuh>
#include <algorithm>
#include <vector>
#include "../../common/cuda_utils.h"
#include "../mxnet_op.h"

#define PSROIPOOLINGALIGN_CUDA_CHECK(condition) \
  /* Code block avoids redefinition of hipError_t error */ \
  do { \
    hipError_t error = condition; \
    CHECK_EQ(error, hipSuccess) << " " << hipGetErrorString(error); \
  } while (0)
#define CUDA_KERNEL_LOOP(i, n) \
for (int i = blockIdx.x * blockDim.x + threadIdx.x; \
      i < (n); \
      i += blockDim.x * gridDim.x)

namespace mshadow {
  namespace cuda {
    template <typename DType>
    __device__ DType bilinear_interp(
      const DType* data,
      const DType x,
      const DType y,
      const int width,
      const int height) {
      int x1 = floor(x);
      int x2 = ceil(x);
      int y1 = floor(y);
      int y2 = ceil(y);
      DType dist_x = static_cast<DType>(x - x1);
      DType dist_y = static_cast<DType>(y - y1);
      DType value11 = data[y1*width + x1];
      DType value12 = data[y2*width + x1];
      DType value21 = data[y1*width + x2];
      DType value22 = data[y2*width + x2];
      DType value = (1 - dist_x)*(1 - dist_y)*value11 + (1 - dist_x)*dist_y*value12
        + dist_x*(1 - dist_y)*value21 + dist_x*dist_y*value22;
      return value;
    }

    template <typename DType>
    __global__ void PSROIPoolAlignForwardKernel(
      const int count,
      const DType* bottom_data,
      const DType spatial_scale,
      const int channels,
      const int height, const int width,
      const int pooled_height, const int pooled_width,
      const DType* bottom_rois,
      const int output_dim,
      const int group_size,
      const int sample_per_part,
      DType* top_data,
      DType* top_count) {
      CUDA_KERNEL_LOOP(index, count) {
        // The output is in order (n, ctop, ph, pw)
        int pw = index % pooled_width;
        int ph = (index / pooled_width) % pooled_height;
        int ctop = (index / pooled_width / pooled_height) % output_dim;
        int n = index / pooled_width / pooled_height / output_dim;

        // [start, end) interval for spatial sampling
        const DType* offset_bottom_rois = bottom_rois + n * 5;
        int roi_batch_ind = offset_bottom_rois[0];
        DType roi_start_w = static_cast<DType>(round(offset_bottom_rois[1])) * spatial_scale - 0.5;
        DType roi_start_h = static_cast<DType>(round(offset_bottom_rois[2])) * spatial_scale - 0.5;
        DType roi_end_w = static_cast<DType>(round(offset_bottom_rois[3]) + 1.) * spatial_scale - 0.5;
        DType roi_end_h = static_cast<DType>(round(offset_bottom_rois[4]) + 1.) * spatial_scale - 0.5;

        // Force too small ROIs to be 1x1
        DType roi_width = max(roi_end_w - roi_start_w, 0.1); //avoid 0
        DType roi_height = max(roi_end_h - roi_start_h, 0.1);

        // Compute w and h at bottom
        DType bin_size_h = roi_height / static_cast<DType>(pooled_height);
        DType bin_size_w = roi_width / static_cast<DType>(pooled_width);
        DType sub_bin_size_h = bin_size_h / static_cast<DType>(sample_per_part);
        DType sub_bin_size_w = bin_size_w / static_cast<DType>(sample_per_part);

        const DType* offset_bottom_data = bottom_data + (roi_batch_ind * channels) * height * width;
        DType wstart = pw*bin_size_w + roi_start_w;
        DType hstart = ph*bin_size_h + roi_start_h;
        DType sum = 0;
        int count = 0;

        int gw = floor(static_cast<DType>(pw)* group_size / pooled_width);
        int gh = floor(static_cast<DType>(ph)* group_size / pooled_height);
        gw = min(max(gw, 0), group_size - 1);
        gh = min(max(gh, 0), group_size - 1);
        for (int ih = 0; ih < sample_per_part; ih++) {
          for (int iw = 0; iw < sample_per_part; iw++) {
            DType w = wstart + iw*sub_bin_size_w;
            DType h = hstart + ih*sub_bin_size_h;
            // bilinear interpolation
            if (w<-0.5 || w>width - 0.5 || h<-0.5 || h>height - 0.5) {
              continue;
            }
            w = min(max(w, 0.), width - 1.);
            h = min(max(h, 0.), height - 1.);
            int c = (ctop*group_size + gh)*group_size + gw;
            DType val = bilinear_interp(offset_bottom_data + c*height*width, w, h, width, height);
            sum += val;
            count++;
          }
        }
        top_data[index] = (count == 0) ? DType(0) : sum / DType(count);
        top_count[index] = count;
      }
    }

    template<typename DType>
    inline void PSROIPoolAlignForward(const Tensor<gpu, 4, DType> &out,
      const Tensor<gpu, 4, DType> &data,
      const Tensor<gpu, 2, DType> &bbox,
      const Tensor<gpu, 4, DType> &top_count,
      const float spatial_scale,
      const int output_dim,
      const int group_size,
      const int sample_per_part) {
      // LOG(INFO) << "PSROIPoolAlignForward";
      const DType *bottom_data = data.dptr_;
      const DType *bottom_rois = bbox.dptr_;
      DType *top_data = out.dptr_;
      DType *top_count_data = top_count.dptr_;
      const int count = out.shape_.Size();
      const int channels = data.size(1);
      const int height = data.size(2);
      const int width = data.size(3);
      const int pooled_height = out.size(2); // in this version, pooled_size == group_size by default
      const int pooled_width = out.size(3);

      hipStream_t stream = Stream<gpu>::GetStream(out.stream_);
      PSROIPoolAlignForwardKernel<DType> << <mxnet::op::mxnet_op::cuda_get_num_blocks(count),
        kBaseThreadNum, 0, stream >> >(
          count, bottom_data, spatial_scale, channels, height, width,
          pooled_height, pooled_width, bottom_rois, output_dim, group_size,
          sample_per_part, top_data, top_count_data);
      PSROIPOOLINGALIGN_CUDA_CHECK(hipPeekAtLastError());
    }


    template <typename DType>
    __global__ void PSROIPoolAlignBackwardAccKernel(
      const int count,
      const DType* top_diff,
      const DType* top_count,
      const int num_rois,
      const DType spatial_scale,
      const int channels,
      const int height, const int width,
      const int pooled_height, const int pooled_width,
      const int output_dim,
      const int group_size,
      const int sample_per_part,
      DType* bottom_diff,
      const DType* bottom_data,
      const DType* bottom_rois) {
      CUDA_KERNEL_LOOP(index, count) {
        // The output is in order (n, ctop, ph, pw)
        int pw = index % pooled_width;
        int ph = (index / pooled_width) % pooled_height;
        int ctop = (index / pooled_width / pooled_height) % output_dim;
        int n = index / pooled_width / pooled_height / output_dim;

        // [start, end) interval for spatial sampling
        const DType* offset_bottom_rois = bottom_rois + n * 5;
        int roi_batch_ind = offset_bottom_rois[0];
        DType roi_start_w = static_cast<DType>(round(offset_bottom_rois[1])) * spatial_scale - 0.5;
        DType roi_start_h = static_cast<DType>(round(offset_bottom_rois[2])) * spatial_scale - 0.5;
        DType roi_end_w = static_cast<DType>(round(offset_bottom_rois[3]) + 1.) * spatial_scale - 0.5;
        DType roi_end_h = static_cast<DType>(round(offset_bottom_rois[4]) + 1.) * spatial_scale - 0.5;

        // Force too small ROIs to be 1x1
        DType roi_width = max(roi_end_w - roi_start_w, 0.1); //avoid 0
        DType roi_height = max(roi_end_h - roi_start_h, 0.1);

        // Compute w and h at bottom
        DType bin_size_h = roi_height / static_cast<DType>(pooled_height);
        DType bin_size_w = roi_width / static_cast<DType>(pooled_width);
        DType sub_bin_size_h = bin_size_h / static_cast<DType>(sample_per_part);
        DType sub_bin_size_w = bin_size_w / static_cast<DType>(sample_per_part);

        DType wstart = pw*bin_size_w + roi_start_w;
        DType hstart = ph*bin_size_h + roi_start_h;
        if (top_count[index] <= 0) {
          continue;
        }
        DType diff_val = top_diff[index] / top_count[index];

        int gw = floor(static_cast<DType>(pw)* group_size / pooled_width);
        int gh = floor(static_cast<DType>(ph)* group_size / pooled_height);
        gw = min(max(gw, 0), group_size - 1);
        gh = min(max(gh, 0), group_size - 1);
        for (int ih = 0; ih < sample_per_part; ih++) {
          for (int iw = 0; iw < sample_per_part; iw++) {
            DType w = wstart + iw*sub_bin_size_w;
            DType h = hstart + ih*sub_bin_size_h;
            // bilinear interpolation
            if (w<-0.5 || w>width - 0.5 || h<-0.5 || h>height - 0.5) {
              continue;
            }
            w = min(max(w, 0.), width - 1.);
            h = min(max(h, 0.), height - 1.);
            int c = (ctop*group_size + gh)*group_size + gw;
            // backward on feature
            int x0 = floor(w);
            int x1 = ceil(w);
            int y0 = floor(h);
            int y1 = ceil(h);
            DType dist_x = w - x0, dist_y = h - y0;
            DType q00 = (1 - dist_x)*(1 - dist_y);
            DType q01 = (1 - dist_x)*dist_y;
            DType q10 = dist_x*(1 - dist_y);
            DType q11 = dist_x*dist_y;
            DType* offset_bottom_data_diff = bottom_diff + roi_batch_ind * channels * height * width + c * height * width;
            atomicAdd(offset_bottom_data_diff + y0*width + x0, q00*diff_val);
            atomicAdd(offset_bottom_data_diff + y1*width + x0, q01*diff_val);
            atomicAdd(offset_bottom_data_diff + y0*width + x1, q10*diff_val);
            atomicAdd(offset_bottom_data_diff + y1*width + x1, q11*diff_val);
          }
        }
      }
    }


    template<typename DType>
    inline void PSROIPoolAlignBackwardAcc(const Tensor<gpu, 4, DType> &in_grad,
      const Tensor<gpu, 4, DType> &out_grad,
      const Tensor<gpu, 4, DType> &data,
      const Tensor<gpu, 2, DType> &bbox,
      const Tensor<gpu, 4, DType> &top_count,
      const float spatial_scale,
      const int output_dim,
      const int group_size,
      const int sample_per_part) {
      // LOG(INFO) << "PSROIPoolAlignBackward";
      const DType *top_diff = out_grad.dptr_;
      const DType *bottom_data = data.dptr_;
      const DType *bottom_rois = bbox.dptr_;
      DType *bottom_diff = in_grad.dptr_;
      DType *top_count_data = top_count.dptr_;
      const int count = out_grad.shape_.Size();
      const int num_rois = bbox.size(0);
      const int channels = in_grad.size(1);
      const int height = in_grad.size(2);
      const int width = in_grad.size(3);
      const int pooled_height = out_grad.size(2);
      const int pooled_width = out_grad.size(3);

      hipStream_t stream = Stream<gpu>::GetStream(in_grad.stream_);
      PSROIPoolAlignBackwardAccKernel<DType> << <mxnet::op::mxnet_op::cuda_get_num_blocks(count),
        kBaseThreadNum, 0, stream >> >(
          count, top_diff, top_count_data, num_rois, spatial_scale, channels, height, width,
          pooled_height, pooled_width, output_dim, group_size, sample_per_part,
          bottom_diff, bottom_data, bottom_rois);
      PSROIPOOLINGALIGN_CUDA_CHECK(hipPeekAtLastError());
    }

  }  // namespace cuda

  template<typename DType>
  inline void PSROIPoolAlignForward(const Tensor<gpu, 4, DType> &out,
    const Tensor<gpu, 4, DType> &data,
    const Tensor<gpu, 2, DType> &bbox,
    const Tensor<gpu, 4, DType> &top_count,
    const float spatial_scale,
    const int output_dim,
    const int group_size,
    const int sample_per_part) {
    cuda::PSROIPoolAlignForward(out, data, bbox, top_count, spatial_scale, output_dim, group_size, sample_per_part);
  }

  template<typename DType>
  inline void PSROIPoolAlignBackwardAcc(const Tensor<gpu, 4, DType> &in_grad,
    const Tensor<gpu, 4, DType> &out_grad,
    const Tensor<gpu, 4, DType> &data,
    const Tensor<gpu, 2, DType> &bbox,
    const Tensor<gpu, 4, DType> &top_count,
    const float spatial_scale,
    const int output_dim,
    const int group_size,
    const int sample_per_part) {
    cuda::PSROIPoolAlignBackwardAcc(in_grad, out_grad, data, bbox, top_count, spatial_scale, output_dim,
      group_size, sample_per_part);
  }

}  // namespace mshadow


namespace mxnet {
  namespace op {

    template<>
    Operator* CreateOp<gpu>(PSROIPoolingAlignParam param, int dtype) {
      Operator* op = NULL;
      MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
        op = new PSROIPoolingAlignOp<gpu, DType>(param);
      });
      return op;
    }

  }  // namespace op
}  // namespace mxnet